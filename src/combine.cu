#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <sstream>
#include <fstream>

#define MAX_DIMS 10
#define TILE 32
#define BASE_THREAD_NUM 32

#define ADD_FUNC       1
#define MUL_FUNC       2
#define ID_FUNC        3
#define NEG_FUNC       4
#define LT_FUNC        5
#define EQ_FUNC        6
#define SIGMOID_FUNC   7
#define RELU_FUNC      8
#define RELU_BACK_FUNC 9
#define LOG_FUNC       10
#define LOG_BACK_FUNC  11
#define EXP_FUNC       12
#define INV_FUNC       13
#define INV_BACK_FUNC  14
#define IS_CLOSE_FUNC  15
#define MAX_FUNC       16
#define POW            17
#define TANH           18

__device__ float fn(int fn_id, float x, float y=0) {
    switch(fn_id) {
      case ADD_FUNC: {
        return x + y;
      }
      case MUL_FUNC: {
        return x * y;
      }
      case ID_FUNC: {
      	return x;
      }
      case NEG_FUNC: {
        return -x;
      }
      case LT_FUNC: {
        if (x < y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case EQ_FUNC: {
        if (x == y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case SIGMOID_FUNC: {
        if (x >= 0) {
          return 1.0 / (1.0 + exp(-x));
        }
        else {
          return exp(x) / (1.0 + exp(x));
        }
      }
      case RELU_FUNC: {
        return max(x, 0.0);
      }
      case RELU_BACK_FUNC: {
        if (x > 0) {
          return y;
        }
        else {
          return 0.0;
        }
      }
      case LOG_FUNC: {
        return log(x + 1e-6);
      }
      case LOG_BACK_FUNC: {
        return y / (x + 1e-6);
      }
      case EXP_FUNC: {
        return exp(x);
      }
      case INV_FUNC: {
        return float(1.0 / x);
      }
      case INV_BACK_FUNC: {
        return -(1.0 / (x * x)) * y;
      }
      case IS_CLOSE_FUNC: {
        return (x - y < 1e-2) && (y - x < 1e-2);
      }
      case MAX_FUNC: {
        if (x > y) {
          return x;
        }
        else {
          return y;
        }
      }
      case POW: {
        return pow(x, y);
      }
      case TANH: {
        return tanh(x);
      }
      default: {
        return x + y;
      }
    }
    
}


__device__ int index_to_position(const int* index, const int* strides, int num_dims) {
    int position = 0;
    for (int i = 0; i < num_dims; ++i) {
        position += index[i] * strides[i];
    }
    return position;
}

__device__ void to_index(int ordinal, const int* shape, int* out_index, int num_dims) {
    int cur_ord = ordinal;
    for (int i = num_dims - 1; i >= 0; --i) {
        int sh = shape[i];
        out_index[i] = cur_ord % sh;
        cur_ord /= sh;
    }
}

__device__ void broadcast_index(const int* big_index, const int* big_shape, const int* shape, int* out_index, int num_dims_big, int num_dims) {
    for (int i = 0; i < num_dims; ++i) {
        if (shape[i] > 1) {
            out_index[i] = big_index[i + (num_dims_big - num_dims)];
        } else {
            out_index[i] = 0;
        }
    }
}


__global__ void MatrixMultiplyKernel(
  float* out,
  const int* out_shape,
  const int* out_strides,
  float* a_storage,
  const int* a_shape,
  const int* a_strides,
  float* b_storage,
  const int* b_shape,
  const int* b_strides
) {
/**
 * Multiply two (compact) matrices into an output (also comapct) matrix. Matrix a and b are both in a batch
 * format, with shape [batch_size, m, n], [batch_size, n, p].
 * Requirements:
 * - All data must be first moved to shared memory.
 * - Only read each cell in a and b once.
 * - Only write to global memory once per kernel.
 * There is guarantee that a_shape[0] == b_shape[0], a_shape[2] == b_shape[1],
 * and out_shape[0] == a_shape[0], out_shape[1] == b_shape[1]
 *
 * Args:
 *   out: compact 1D array of size batch_size x m x p to write the output to
 *   out_shape: shape of the output array
 *   out_strides: strides of the output array
 *   a_storage: compact 1D array of size batch_size x m x n
 *   a_shape: shape of the a array
 *   a_strides: strides of the a array
 *   b_storage: comapct 2D array of size batch_size x n x p
 *   b_shape: shape of the b array
 *   b_strides: strides of the b array
 *
 * Returns:
 *   None (Fills in out array)
 */

  __shared__ float a_shared[TILE][TILE];
  __shared__ float b_shared[TILE][TILE];

  // In each block, we will compute a batch of the output matrix
  // All the threads in the block will work together to compute this batch
  int batch = blockIdx.z;
  int a_batch_stride = a_shape[0] > 1 ? a_strides[0] : 0; 
  int b_batch_stride = b_shape[0] > 1 ? b_strides[0] : 0;


  /// BEGIN ASSIGN1_2
  /// TODO
  // Hints:
  // 1. Compute the row and column of the output matrix this block will compute
  // 2. Compute the position in the output array that this thread will write to
  // 3. Iterate over tiles of the two input matrices, read the data into shared memory
  // 4. Synchronize to make sure the data is available to all threads
  // 5. Compute the output tile for this thread block
  // 6. Synchronize to make sure all threads are done computing the output tile for (row, col)
  // 7. Write the output to global memory

  // step 1
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  float sum = 0.0f;

  int out_idx[MAX_DIMS] = {batch, row, col};
  int a_index[MAX_DIMS];
  int b_index[MAX_DIMS];

  // step 2
  int out_pos = index_to_position(out_idx, out_strides, 3);

  // step 3
  for (int k = 0; k < (a_shape[2] + TILE - 1) / TILE; k++) {
    a_index[0] = batch, a_index[1] = row, a_index[2] = k * TILE + threadIdx.y;
    b_index[0] = batch, b_index[1] = k * TILE + threadIdx.x, b_index[2] = col;

    // check if the index is within the bounds of the matrix
    if (a_index[1] < a_shape[1] && a_index[2] < a_shape[2]) {
      a_shared[threadIdx.x][threadIdx.y] = a_storage[index_to_position(a_index, a_strides, 3)];
    } else {
      a_shared[threadIdx.x][threadIdx.y] = 0.0f;
    }

    if (b_index[1] < a_shape[2] && b_index[2] < b_shape[2]) {
      b_shared[threadIdx.x][threadIdx.y] = b_storage[index_to_position(b_index, b_strides, 3)];
    } else {
      b_shared[threadIdx.x][threadIdx.y] = 0.0f;
    }

    // step 4
    __syncthreads();

    // step 5
    for (int i = 0; i < TILE; ++i) {
      if ((k + i) < a_shape[2]) sum += a_shared[threadIdx.x][i] * b_shared[i][threadIdx.y];
    }

    // step 6
    __syncthreads();
  
  }

  // step 7
  if (row < out_shape[1] && col < out_shape[2]) out[out_pos] = sum;

  // assert(false && "Not Implemented");
  /// END ASSIGN1_2
}


__global__ void mapKernel(
  float* out, 
  int* out_shape, 
  int* out_strides, 
  int out_size, 
  float* in_storage, 
  int* in_shape, 
  int* in_strides,
  int shape_size,
  int fn_id
) {
/**
 * Map function. Apply a unary function to each element of the input array and store the result in the output array.
 * Optimization: Parallelize over the elements of the output array.
 *
 * You may find the following functions useful:
 * - index_to_position: converts an index to a position in a compact array
 * - to_index: converts a position to an index in a multidimensional array
 * - broadcast_index: converts an index in a smaller array to an index in a larger array
 *
 * Args:
 *  out: compact 1D array of size out_size to write the output to
 *  out_shape: shape of the output array
 *  out_strides: strides of the output array
 *  out_size: size of the output array
 *  in_storage: compact 1D array of size in_size
 *  in_shape: shape of the input array
 *  in_strides: strides of the input array
 *  shape_size: number of dimensions in the input and output arrays, assume dimensions are the same
 *  fn_id: id of the function to apply to each element of the input array
 *
 * Returns:
 *  None (Fills in out array)
 */

  int out_index[MAX_DIMS];
  int in_index[MAX_DIMS];
  
  /// BEGIN ASSIGN1_2
  /// TODO
  // Hints:
  // 1. Compute the position in the output array that this thread will write to
  // 2. Convert the position to the out_index according to out_shape
  // 3. Broadcast the out_index to the in_index according to in_shape (optional in some cases)
  // 4. Calculate the position of element in in_array according to in_index and in_strides
  // 5. Calculate the position of element in out_array according to out_index and out_strides
  // 6. Apply the unary function to the input element and write the output to the out memory

  // step 1
  int pos = blockIdx.x * blockDim.x + threadIdx.x;

  if (pos < out_size) {
    // step 2
    to_index(pos, out_shape, out_index, shape_size);

    // step 3
    broadcast_index(out_index, out_shape, in_shape, in_index, shape_size, shape_size);

    // step 4
    int in_pos = index_to_position(in_index, in_strides, shape_size);

    // step 5
    int out_pos = index_to_position(out_index, out_strides, shape_size);

    // step 6
    out[out_pos] = fn(fn_id, in_storage[in_pos]);

  }
  
  // assert(false && "Not Implemented");
  /// END ASSIGN1_2
}


__global__ void reduceKernel(
  float* out,
  int* out_shape,
  int* out_strides,
  int out_size,
  float* a_storage,
  int* a_shape,
  int* a_strides,
  int reduce_dim,
  float reduce_value,
  int shape_size,
  int fn_id
) {
/**
 * Reduce function. Apply a reduce function to elements of the input array a and store the result in the output array.
 * Optimization:
 * Parallelize over the reduction operation. Each kernel performs one reduction.
 * e.g. a = [[1, 2, 3], [4, 5, 6]], kernel0 computes reduce([1, 2, 3]), kernel1 computes reduce([4, 5, 6]).
 *
 * You may find the following functions useful:
 * - index_to_position: converts an index to a position in a compact array
 * - to_index: converts a position to an index in a multidimensional array
 *
 * Args:
 *  out: compact 1D array of size out_size to write the output to
 *  out_shape: shape of the output array
 *  out_strides: strides of the output array
 *  out_size: size of the output array
 *  a_storage: compact 1D array of size in_size
 *  a_shape: shape of the input array
 *  a_strides: strides of the input array
 *  reduce_dim: dimension to reduce on
 *  reduce_value: initial value for the reduction
 *  shape_size: number of dimensions in the input & output array, assert dimensions are the same
 *  fn_id: id of the reduce function, currently only support add, multiply, and max
 *
 *
 * Returns:
 *  None (Fills in out array)
 */

  // __shared__ double cache[BLOCK_DIM]; // Uncomment this line if you want to use shared memory to store partial results
  int out_index[MAX_DIMS];

  /// BEGIN ASSIGN1_2
  /// TODO
  // 1. Define the position of the output element that this thread or this block will write to
  // 2. Convert the out_pos to the out_index according to out_shape
  // 3. Initialize the reduce_value to the output element
  // 4. Iterate over the reduce_dim dimension of the input array to compute the reduced value
  // 5. Write the reduced value to out memory

  // step 1
  int out_position = blockIdx.x * blockDim.x + threadIdx.x;

  if (out_position < out_size) {
    // step 2
    to_index(out_position, out_shape, out_index, shape_size);

    // step 3
    float reduced_value = reduce_value;

    // step 4
    for (int i = 0; i < a_shape[reduce_dim]; ++i) {
      int a_index[MAX_DIMS];

      for (int j = 0; j < shape_size; ++j) {
        a_index[j] = out_index[j];
      }
      a_index[reduce_dim] = i; // reduce along dimension of reduce_dim

      // convert a_index to a_pos to get the position of the element in a_storage
      int a_pos = index_to_position(a_index, a_strides, shape_size); 

      // apply the reduce function to the element
      reduced_value = fn(fn_id, reduced_value, a_storage[a_pos]);

    }

    // step 5
    out[out_position] = reduced_value;
  }
  
  // assert(false && "Not Implemented");
  /// END ASSIGN1_2
}

__global__ void zipKernel(
  float* out,
  int* out_shape,
  int* out_strides,
  int out_size,
  int out_shape_size,
  float* a_storage,
  int* a_shape,
  int* a_strides,
  int a_shape_size,
  float* b_storage, 
  int* b_shape, 
  int* b_strides,
  int b_shape_size,
  int fn_id
) {
/**
 * Zip function. Apply a binary function to elements of the input array a & b and store the result in the output array.
 * Optimization: Parallelize over the elements of the output array.
 *
 * You may find the following functions useful:
 * - index_to_position: converts an index to a position in a compact array
 * - to_index: converts a position to an index in a multidimensional array
 * - broadcast_index: converts an index in a smaller array to an index in a larger array
 *
 * Args:
 *  out: compact 1D array of size out_size to write the output to
 *  out_shape: shape of the output array
 *  out_strides: strides of the output array
 *  out_size: size of the output array
 *  out_shape_size: number of dimensions in the output array
 *  a_storage: compact 1D array of size in_size
 *  a_shape: shape of the input array
 *  a_strides: strides of the input array
 *  a_shape_size: number of dimensions in the input array
 *  b_storage: compact 1D array of size in_size
 *  b_shape: shape of the input array
 *  b_strides: strides of the input array
 *  b_shape_size: number of dimensions in the input array
 *  fn_id: id of the function to apply to each element of the a & b array
 *
 *
 * Returns:
 *  None (Fills in out array)
 */

  int out_index[MAX_DIMS];
  int a_index[MAX_DIMS];
  int b_index[MAX_DIMS];

  /// BEGIN ASSIGN1_2
  /// TODO
  // Hints:
  // 1. Compute the position in the output array that this thread will write to
  // 2. Convert the position to the out_index according to out_shape
  // 3. Calculate the position of element in out_array according to out_index and out_strides
  // 4. Broadcast the out_index to the a_index according to a_shape
  // 5. Calculate the position of element in a_array according to a_index and a_strides
  // 6. Broadcast the out_index to the b_index according to b_shape
  // 7.Calculate the position of element in b_array according to b_index and b_strides
  // 8. Apply the binary function to the input elements in a_array & b_array and write the output to the out memory
  
  // step 1
  int pos = blockIdx.x * blockDim.x + threadIdx.x;

  if (pos < out_size) {
    // step 2
    to_index(pos, out_shape, out_index, out_shape_size);

    // step 3
    int out_pos = index_to_position(out_index, out_strides, out_shape_size);

    // step 4
    broadcast_index(out_index, out_shape, a_shape, a_index, out_shape_size, a_shape_size);

    // step 5
    int a_pos = index_to_position(a_index, a_strides, a_shape_size);

    // step 6
    broadcast_index(out_index, out_shape, b_shape, b_index, out_shape_size, b_shape_size);

    // step 7
    int b_pos = index_to_position(b_index, b_strides, b_shape_size);

    // step 8
    out[pos] = fn(fn_id, a_storage[a_pos], b_storage[b_pos]);


  }

  // assert(false && "Not Implemented");
  /// END ASSIGN1_2
}


extern "C" {

void MatrixMultiply(
    float* out,
    int* out_shape,
    int* out_strides,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int batch, int m, int p
) {
    int n = a_shape[2];

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc(&d_a, batch * m * n * sizeof(float));
    hipMalloc(&d_b, batch * n * p * sizeof(float));
    hipMalloc(&d_out, batch * m * p * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, 3 * sizeof(int));
    hipMalloc(&d_out_strides, 3 * sizeof(int));
    hipMalloc(&d_a_shape, 3 * sizeof(int));
    hipMalloc(&d_a_strides, 3 * sizeof(int));
    hipMalloc(&d_b_shape, 3 * sizeof(int));
    hipMalloc(&d_b_strides, 3 * sizeof(int));


    // Copy data to the device
    hipMemcpy(d_a, a_storage, batch * m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, batch * n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, 3 * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = BASE_THREAD_NUM;
    dim3 blockDims(threadsPerBlock, threadsPerBlock, 1); // Adjust these values based on your specific requirements
    dim3 gridDims((m + threadsPerBlock - 1) / threadsPerBlock, (p + threadsPerBlock - 1) / threadsPerBlock, batch);
    MatrixMultiplyKernel<<<gridDims, blockDims>>>(
        d_out, d_out_shape, d_out_strides, d_a, d_a_shape, d_a_strides, d_b, d_b_shape, d_b_strides
    );

    // Copy back to the host
    hipMemcpy(out, d_out, batch * m * p * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Matmul Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}

void tensorMap(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int in_size,
    int shape_size,
    int fn_id
) {

    float *d_out, *d_in;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_in, in_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_in_shape, *d_in_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_in_shape, shape_size * sizeof(int));
    hipMalloc(&d_in_strides, shape_size * sizeof(int));

    hipMemcpy(d_in, in_storage, in_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_shape, in_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_strides, in_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    mapKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, 
      d_in, d_in_shape, d_in_strides, 
      shape_size, fn_id);
    
    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Map Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_in_shape);
    hipFree(d_in_strides);
}


void tensorZip(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_size,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_size,
    int b_shape_size,
    int fn_id
) {

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc((void **)&d_a, a_size * sizeof(float));
    hipMalloc(&d_b, b_size * sizeof(float));
    hipMalloc(&d_out, out_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, out_shape_size * sizeof(int));
    hipMalloc(&d_out_strides, out_shape_size * sizeof(int));
    hipMalloc(&d_a_shape, a_shape_size * sizeof(int));
    hipMalloc(&d_a_strides, a_shape_size * sizeof(int));
    hipMalloc(&d_b_shape, b_shape_size * sizeof(int));
    hipMalloc(&d_b_strides, b_shape_size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, b_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, b_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, b_shape_size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    zipKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, out_shape_size,
      d_a, d_a_shape, d_a_strides, a_shape_size,
      d_b, d_b_shape, d_b_strides, b_shape_size,
      fn_id);

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();


    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Zip Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}



void tensorReduce(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim, 
    float reduce_value,
    int shape_size,
    int fn_id
) {
    int a_size = out_size * a_shape[reduce_dim];
    float *d_out, *d_a;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_a, a_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_a_shape, shape_size * sizeof(int));
    hipMalloc(&d_a_strides, shape_size * sizeof(int));

    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    reduceKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_out, d_out_shape, d_out_strides, out_size, 
        d_a, d_a_shape, d_a_strides, 
        reduce_dim, reduce_value, shape_size, fn_id
    );

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Reduce Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
}

}